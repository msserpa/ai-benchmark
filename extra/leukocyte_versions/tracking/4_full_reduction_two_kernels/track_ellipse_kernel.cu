#include "hip/hip_runtime.h"
#include "track_ellipse_kernel.h"
#include <cutil.h>
#include <sys/time.h>
#include <time.h>

#define ONE_OVER_PI 1.0 / PI
#define MU 0.5
#define LAMBDA (8.0 * MU + 1.0)


float *host_I, *host_IMGVF;
float *device_I, *device_IMGVF_in, *device_IMGVF_out;
float *device_partial_sums, *host_partial_sums;
int *device_converged, *device_in_out;

const int threads_per_block = 64;

__global__ void IMGVF_kernel(float *IMGVF_in, float *IMGVF_out, float *I, float *device_partial_sums, int *converged, float vx, float vy, float e, int m, int n) {
	if (*converged == 1) return;

	// Determine the thread's coordinates
	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = thread_id / n;
	int j = thread_id % n;
	
	float new_val = 0.0, old_val = 0.0;
	
	if (i < m) {
		// Compute neighboring pixel indices
		int rowU = (i == 0) ? 0 : i - 1;
		int rowD = (i == m - 1) ? m - 1 : i + 1;
		int colL = (j == 0) ? 0 : j - 1;
		int colR = (j == n - 1) ? n - 1 : j + 1;
		
		// Compute the difference between the pixel and its eight neighbors
		old_val = IMGVF_in[(i * n) + j];
		float U  = IMGVF_in[(rowU * n) + j   ] - old_val;
		float D  = IMGVF_in[(rowD * n) + j   ] - old_val;
		float L  = IMGVF_in[(i    * n) + colL] - old_val;
		float R  = IMGVF_in[(i    * n) + colR] - old_val;
		float UR = IMGVF_in[(rowU * n) + colR] - old_val;
		float DR = IMGVF_in[(rowD * n) + colR] - old_val;
		float UL = IMGVF_in[(rowU * n) + colL] - old_val;
		float DL = IMGVF_in[(rowD * n) + colL] - old_val;
		
		// Compute the regularized heaviside value for these differences
		float one_over_e = 1.0 / e;
		float UHe  = ONE_OVER_PI * atan((U  *       -vy)  * one_over_e) + 0.5;
		float DHe  = ONE_OVER_PI * atan((D  *        vy)  * one_over_e) + 0.5;
		float LHe  = ONE_OVER_PI * atan((L  *  -vx     )  * one_over_e) + 0.5;
		float RHe  = ONE_OVER_PI * atan((R  *   vx     )  * one_over_e) + 0.5;
		float URHe = ONE_OVER_PI * atan((UR * ( vx - vy)) * one_over_e) + 0.5;
		float DRHe = ONE_OVER_PI * atan((DR * ( vx + vy)) * one_over_e) + 0.5;
		float ULHe = ONE_OVER_PI * atan((UL * (-vx - vy)) * one_over_e) + 0.5;
		float DLHe = ONE_OVER_PI * atan((DL * (-vx + vy)) * one_over_e) + 0.5;
		
		// Update the IMGVF value
		// Compute IMGVF += (mu / lambda)(UHe .*U  + DHe .*D  + LHe .*L  + RHe .*R +
		//                                URHe.*UR + DRHe.*DR + ULHe.*UL + DLHe.*DL);
		new_val = old_val + (MU / LAMBDA) * (UHe  * U  + DHe  * D  + LHe  * L  + RHe  * R +
											 URHe * UR + DRHe * DR + ULHe * UL + DLHe * DL);
		// Compute IMGVF -= (1 / lambda)(I .* (IMGVF - I))
		float vI = I[(i * n) + j];
		new_val -= ((1.0 / LAMBDA) * vI * (new_val - vI));
		IMGVF_out[(i * n) + j] = new_val;
	}
	
	// Sum the absolute values of the differences
	//  across the entire thread block
	__shared__ float val[threads_per_block];
	val[threadIdx.x] = fabs(new_val - old_val);
	__syncthreads();
	
	// Perform the reduction
	int th;
	for (th = threads_per_block / 2; th > 0; th /= 2) {
		if (threadIdx.x < th) val[threadIdx.x] += val[threadIdx.x + th];
		__syncthreads();
	}
	
	// Save the final value
	if (threadIdx.x == 0) device_partial_sums[blockIdx.x] = val[0];
}


__global__ void reduce_kernel(float *partial_sums, int num_blocks, int num_threads, float converge, int *converged, int *in_out) {
	if (*converged == 1) return;

	__shared__ float val[256];
	val[threadIdx.x] = partial_sums[threadIdx.x];
	
	if (threadIdx.x == 0) {
		*in_out = 1 - *in_out;
		int i;
		float sum = 0.0;
		for (i = 0; i < num_blocks; i++) sum += val[i];
		float mean = sum / (float) num_threads;
		if (mean < converge) *converged = 1;
	}
}


extern "C" void IMGVF_cuda(MAT *I, MAT *IMGVF, double vx, double vy, double e, int iterations, double cutoff);

// Note: about 2/3 of the execution time is spent in the kernel
//       the other 1/3 is spent copying memory back and forth
void IMGVF_cuda(MAT *I, MAT *IMGVF, double vx, double vy, double e, int iterations, double cutoff) {
	
	// Initialize the data on the GPU
	IMGVF_cuda_init(I);

	// Determine thread block size
	int m = IMGVF->m, n = IMGVF->n;
	int num_threads = m * n;
	int num_blocks = (int) (((float) num_threads / (float) threads_per_block) + 0.5);
    
	// struct timeval tv;
    // gettimeofday(&tv, NULL); 
    // long long loop_start_time = tv.tv_sec*1000000 + tv.tv_usec;
	// long long copy_time = 0;
	
	// Compute the MGVF
	int host_converged = 0, iter = 0;
	while ((iter < iterations) && (! host_converged)) {
		// Execute the kernel multiple times
		int i, unroll = 16;
		for (i = 0; i < unroll; i++) {
			IMGVF_kernel <<< num_blocks, threads_per_block >>> (device_IMGVF_in, device_IMGVF_out, device_I, device_partial_sums, device_converged, (float) vx, (float) vy, (float) e, m, n);
			reduce_kernel <<< 1, num_blocks >>> (device_partial_sums, num_blocks, num_threads, (float) cutoff, device_converged, device_in_out);
			
			IMGVF_kernel <<< num_blocks, threads_per_block >>> (device_IMGVF_out, device_IMGVF_in, device_I, device_partial_sums, device_converged, (float) vx, (float) vy, (float) e, m, n);
			reduce_kernel <<< 1, num_blocks >>> (device_partial_sums, num_blocks, num_threads, (float) cutoff, device_converged, device_in_out);
		}
		
		// hipDeviceSynchronize();
		// gettimeofday(&tv, NULL); long long copy_start_time = tv.tv_sec*1000000 + tv.tv_usec;
		hipMemcpy(&host_converged, device_converged, sizeof(int), hipMemcpyDeviceToHost);
		// gettimeofday(&tv, NULL); long long copy_end_time = tv.tv_sec*1000000 + tv.tv_usec;
		// copy_time += copy_end_time - copy_start_time;
		
		iter += (unroll * 2);
	}
	
	// gettimeofday(&tv, NULL); 
    // long long loop_end_time = tv.tv_sec*1000000 + tv.tv_usec;
	// printf("kernel,copy:");
	// printf(" %.8f,", ((float) (loop_end_time - loop_start_time - copy_time)) / (1000*1000));
	// printf(" %.8f\n", ((float) (copy_time)) / (1000*1000));
	
	// Copy back the final results from the GPU
	IMGVF_cuda_cleanup(IMGVF);
}


extern "C" void IMGVF_cuda_init(MAT *I);

void IMGVF_cuda_init(MAT *I) {
	// Compute the amount of memory required and the number of thread blocks
	int m = I->m, n = I->n, i, j;
	int mem_size = sizeof(float) * m * n;
	int num_threads = m * n;
	int num_blocks = (int) (((float) num_threads / (float) threads_per_block) + 0.5);
	
	//Allocate device memory
	hipMalloc( (void**) &device_IMGVF_in, mem_size);
	hipMalloc( (void**) &device_IMGVF_out, mem_size);
	hipMalloc( (void**) &device_I, mem_size);
	hipMalloc( (void**) &device_partial_sums, num_blocks*sizeof(float));
	hipMalloc( (void**) &device_converged, sizeof(int));
	hipMalloc( (void**) &device_in_out, sizeof(int));
	
	// Allocate host memory
	host_IMGVF = (float *) malloc(mem_size);
	host_I = (float *) malloc(mem_size);
	host_partial_sums = (float *) malloc(num_blocks*sizeof(float));
	
	// Copy matrix I (which is also the initial IMGVF matrix) to device
	for (i = 0; i < m; i++)
		for (j = 0; j < n; j++)
			host_I[(i * n) + j] = (float) m_get_val(I, i, j);
	hipMemcpy(device_I, host_I, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(device_IMGVF_in, host_I, mem_size, hipMemcpyHostToDevice);
	
	// Set up convergence variables
	int boolean = 0;
	hipMemcpy(device_converged, &boolean, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_in_out, &boolean, sizeof(int), hipMemcpyHostToDevice);
}


extern "C" void IMGVF_cuda_cleanup(MAT *IMGVF_out);

void IMGVF_cuda_cleanup(MAT *IMGVF_out) {
	// Compute the amount of memory required
	int m = IMGVF_out->m, n = IMGVF_out->n, i, j;
	int mem_size = sizeof(float) * m * n;
	
	// Determine which array to copy from
	int host_in_out;
	hipMemcpy(&host_in_out, device_in_out, sizeof(int), hipMemcpyDeviceToHost);
	float *IMGVF;
	if (host_in_out) IMGVF = device_IMGVF_out;
	else             IMGVF = device_IMGVF_in;
	
	// Copy result from device to host
	hipMemcpy(host_IMGVF, IMGVF, mem_size, hipMemcpyDeviceToHost);
	// Pack the result into the matrix
	for (i = 0; i < m; i++)
		for (j = 0; j < n; j++)
			m_set_val(IMGVF_out, i, j, (double) host_IMGVF[(i * n) + j]);

	// Free memory
	free(host_IMGVF);
	free(host_I);
	free(host_partial_sums);
	hipFree(device_IMGVF_in);
	hipFree(device_IMGVF_out);
	hipFree(device_I);
	hipFree(device_partial_sums);
	hipFree(device_converged);
	hipFree(device_in_out);
}
