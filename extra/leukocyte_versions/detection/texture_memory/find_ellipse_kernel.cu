#include "hip/hip_runtime.h"
//#ifndef _FIND_ELLIPSE_KERNEL_H_
//#define _FIND_ELLIPSE_KERNEL_H_

#include <cutil.h>
#include "find_ellipse_kernel.h"

#include <stdio.h>

#define NPOINTS 150
#define MIN_RAD 10
#define MAX_RAD 20
#define NCIRCLES 10
#define NCIRC 7


__constant__ float c_sin_angle[NPOINTS];
__constant__ float c_cos_angle[NPOINTS];
__constant__ int c_tX[NCIRCLES * NPOINTS];
__constant__ int c_tY[NCIRCLES * NPOINTS];


#define STREL_SIZE (12 * 2 + 1)
__constant__ float c_strel[STREL_SIZE * STREL_SIZE];

// Texture references
texture<float, 1, hipReadModeElementType> t_grad_x;
texture<float, 1, hipReadModeElementType> t_grad_y;
texture<float, 1, hipReadModeElementType> t_img;


//Find matrix of GICOV values at all given pixels given x and y gradients of the image
__global__ void ellipsematching_kernel(float * grad_x, float * grad_y, int grad_n, float * sin_angle, float * cos_angle, int * tX, int * tY, float * gicov) {
	float Grad[NPOINTS];
	float sum, ep, ave, var, sGicov;
	int i, j, k, n, x, y;

	i = blockIdx.x + MAX_RAD + 2;
	j = threadIdx.x + (threadIdx.y * blockDim.x) + (threadIdx.z * blockDim.x * blockDim.y) + MAX_RAD + 2;

	sGicov = 0;
			
	for(k = 0; k < NCIRC; k++) {
		for(n = 0; n < NPOINTS; n++) {
			y = j + c_tY[(k * NPOINTS) + n];
			x = i + c_tX[(k * NPOINTS) + n];
			int addr = (y * grad_n) + x;
			Grad[n] = tex1Dfetch(t_grad_x,addr) * c_cos_angle[n] + tex1Dfetch(t_grad_y,addr) * c_sin_angle[n];
		}
		
		sum = 0.0;
		ep = 0.0;

		for(n = 0; n < NPOINTS; n++) sum += Grad[n];

		ave = sum / ((float) NPOINTS);
		var = 0.0;

		for(n = 0; n < NPOINTS; n++) {
			sum = Grad[n] - ave;
			var += sum * sum;
			ep += sum;
		}

		var = (var - ((ep * ep)/((float) NPOINTS))) / ((float) (NPOINTS - 1));

		if(((ave * ave) / var) > sGicov) {
			gicov[(j * grad_n) + i] = ave / sqrt(var);
			sGicov = (ave * ave) / var;
		}
	}
}



extern "C" float *ellipsematching_setup(int grad_m, int grad_n, float *host_grad_x, float *host_grad_y);

float *ellipsematching_setup(int grad_m, int grad_n, float *host_grad_x, float *host_grad_y) {

	int MaxR = MAX_RAD + 2;

	unsigned int grad_mem_size = sizeof(float) * grad_m * grad_n;

	// allocate device memory
	float *device_grad_x, *device_grad_y;
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_grad_x, grad_mem_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_grad_y, grad_mem_size));

	// copy host memory to device
	CUDA_SAFE_CALL( hipMemcpy( device_grad_x, host_grad_x, grad_mem_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( device_grad_y, host_grad_y, grad_mem_size, hipMemcpyHostToDevice) );

	// Bind input matrices to textures
	CUDA_SAFE_CALL( hipBindTexture(0, t_grad_x, device_grad_x, grad_mem_size));
	CUDA_SAFE_CALL( hipBindTexture(0, t_grad_y, device_grad_y, grad_mem_size));

	// allocate mem for the result on host side
	float *host_gicov = (float *) malloc(grad_mem_size);
	// initialize memory (some of the cells are never assigned a value in the kernel)
	for (int i = 0; i < grad_m * grad_n; i++) host_gicov[i] = 0.0;
	// allocate device memory for result
	float *device_gicov;
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_gicov, grad_mem_size));
	// copy zeroed host memory to device
	CUDA_SAFE_CALL( hipMemcpy( device_gicov, host_gicov, grad_mem_size, hipMemcpyHostToDevice) );

	// setup execution parameters
	dim3 grid( grad_n - (2 * MaxR), 1, 1);
	dim3 threads( (grad_m - (2 * MaxR)) / 7 / 5, 7, 5);

	// execute the kernel
	ellipsematching_kernel<<< grid, threads, 0 >>>(device_grad_x, device_grad_y, grad_n, NULL, NULL, NULL, NULL, device_gicov);

	// check if kernel execution caused an error
	CUT_CHECK_ERROR("Kernel execution failed");

	// copy result from device to host
	CUDA_SAFE_CALL( hipMemcpy( host_gicov, device_gicov, grad_mem_size, hipMemcpyDeviceToHost) );

	// cleanup memory
	free(host_grad_x);
	free(host_grad_y);
	hipUnbindTexture(t_grad_x);
	hipUnbindTexture(t_grad_y);
	CUDA_SAFE_CALL(hipFree(device_grad_x));
	CUDA_SAFE_CALL(hipFree(device_grad_y));
	CUDA_SAFE_CALL(hipFree(device_gicov));

	return host_gicov;
}




__global__ void dilate_f_kernel(float *img, int img_m, int img_n, float *strel, int strel_m, int strel_n, float *dilated) {
	int i, j, el_i, el_j, x, y;
	int el_center_i = strel_m / 2, el_center_j = strel_n / 2;
	float max, temp;

	i = blockIdx.y;
	j = (blockIdx.x * blockDim.x) + threadIdx.x;

	max = 0.0;
	
	for(el_i = 0; el_i < strel_m; el_i++) {
		y = i - el_center_i + el_i;
		if((y >= 0) && (y < img_m)) {
			for(el_j = 0; el_j < strel_n; el_j++) {
				x = j - el_center_j + el_j;
				if ((x >= 0) &&	(x < img_n) && (c_strel[(el_i * strel_n) + el_j] != 0)) {
					int addr = (y * img_n) + x;
					temp = tex1Dfetch(t_img, addr);
					if (temp > max) max = temp;
				}
			}
		}
	}

	dilated[(i * img_n) + j] = max;
}



extern "C" float *dilate_f_setup(int max_gicov_m, int max_gicov_n, int strel_m, int strel_n, float *host_max_gicov);

float *dilate_f_setup(int max_gicov_m, int max_gicov_n, int strel_m, int strel_n, float *host_max_gicov) {

  // Compute memory sizes
  unsigned int max_gicov_mem_size = sizeof(float) * max_gicov_m * max_gicov_n;

  // allocate device memory
  float *device_max_gicov;
  CUDA_SAFE_CALL( hipMalloc( (void**) &device_max_gicov, max_gicov_mem_size));
  
  // copy host memory to device
  CUDA_SAFE_CALL( hipMemcpy( device_max_gicov, host_max_gicov, max_gicov_mem_size, hipMemcpyHostToDevice) );

  // Bind matrix to texture
  CUDA_SAFE_CALL( hipBindTexture(0, t_img, device_max_gicov, max_gicov_mem_size));
  
  // allocate device memory for result
  float* device_img_dilated;
  CUDA_SAFE_CALL( hipMalloc( (void**) &device_img_dilated, max_gicov_mem_size));
  
  // setup execution parameters
  dim3 grid( 2, max_gicov_m, 1);
  dim3 threads( max_gicov_n / 2, 1, 1);
  
//printf("Grid: %d x %d x %d\n", grid.x, grid.y, grid.z);
//printf("Threads: %d x %d x %d\n", threads.x, threads.y, threads.z);
  
  // execute the kernel
  dilate_f_kernel<<< grid, threads, 0 >>>(device_max_gicov, max_gicov_m, max_gicov_n, NULL, strel_m, strel_n, device_img_dilated);
  
  // check if kernel execution generated an error
  CUT_CHECK_ERROR("Kernel execution failed");
  
  // allocate mem for the result on host side
  float *host_img_dilated = (float*) malloc(max_gicov_mem_size);
  // copy result from device to host
  CUDA_SAFE_CALL(hipMemcpy(host_img_dilated, device_img_dilated, max_gicov_mem_size, hipMemcpyDeviceToHost));
  
  //printf("dilate_f_kernel: %s\n", hipGetErrorString(hipGetLastError()));
  
  // cleanup memory
  hipUnbindTexture(t_img);
  free(host_max_gicov);
  CUDA_SAFE_CALL(hipFree(device_max_gicov));
  CUDA_SAFE_CALL(hipFree(device_img_dilated));
  
  return host_img_dilated;
}


// Chooses the most appropriate GPU on which to execute
void select_device() {
	// Figure out how many devices exist
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	
	// Choose the device with the largest number of multiprocessors
	if (num_devices > 0) {
		int max_multiprocessors = 0, max_device = -1;
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
	}
	
	// The following is to remove the API initialization overhead from the runtime measurements
	hipFree(0);
}


void transfer_constants(float *host_sin_angle, float *host_cos_angle, int *host_tX, int *host_tY, int strel_m, int strel_n, float *host_strel) {

	// Compute the sizes of the matrices
	unsigned int angle_mem_size = sizeof(float) * NPOINTS;
	unsigned int t_mem_size = sizeof(int) * NCIRCLES * NPOINTS;
	unsigned int strel_mem_size = sizeof(float) * strel_m * strel_n;

	// Copy the matrices from host memory to device constant memory
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_sin_angle"), host_sin_angle, angle_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_cos_angle"), host_cos_angle, angle_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_tX"), host_tX, t_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_tY"), host_tY, t_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_strel"), host_strel, strel_mem_size, 0, hipMemcpyHostToDevice));
}

