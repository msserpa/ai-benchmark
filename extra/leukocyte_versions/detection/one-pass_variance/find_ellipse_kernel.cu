#include "hip/hip_runtime.h"
#include <cutil.h>
#include "find_ellipse_kernel.h"

#include <stdio.h>
#include <sys/param.h>
#include <sys/times.h>

#define NPOINTS 150
#define MIN_RAD 10
#define MAX_RAD 20
#define NCIRCLES 10
#define NCIRC 7

#define STREL_SIZE (12 * 2 + 1)

float *device_gicov;

__constant__ float c_sin_angle[NPOINTS];
__constant__ float c_cos_angle[NPOINTS];
__constant__ int c_tX[NCIRCLES * NPOINTS];
__constant__ int c_tY[NCIRCLES * NPOINTS];

__constant__ float c_strel[STREL_SIZE * STREL_SIZE];

// Texture references
texture<float, 1, hipReadModeElementType> t_grad_x;
texture<float, 1, hipReadModeElementType> t_grad_y;
texture<float, 1, hipReadModeElementType> t_img;

//Find matrix of GICOV values at all given pixels given x and y gradients of the image
__global__ void ellipsematching_kernel(float * grad_x, float * grad_y, int grad_m, int grad_n, float * gicov) {
	float sum, ave, var, sGicov;
	int i, j, k, n, x, y;
    
	i = blockIdx.x + MAX_RAD + 2;
	j = threadIdx.x + MAX_RAD + 2;

	sGicov = 0;

	for (k = 0; k < NCIRC; k++) {
		sum = 0.0;
		float M2 = 0.f;
		float mean = 0.f;
	
		for (n = 0; n < NPOINTS; n++) {
			y = j + c_tY[(k * NPOINTS) + n];
			x = i + c_tX[(k * NPOINTS) + n];
			int addr = x * grad_m + y;
			float p =   tex1Dfetch(t_grad_x,addr) * c_cos_angle[n] + 
						tex1Dfetch(t_grad_y,addr) * c_sin_angle[n];                    
			sum += p;
			float delta = p - mean;
			mean = mean + (delta / (float) (n + 1));
			M2 = M2 + (delta * (p - mean));
		}
		
		ave = sum / ((float) NPOINTS);
		
		var = M2 / ((float) (NPOINTS - 1));
		
		if (((ave * ave) / var) > sGicov) {
			gicov[(i * grad_m) + j] = ave / sqrt(var);
			sGicov = (ave * ave) / var;
		}
	}
	
	// Square the gicov value
	gicov[(i * grad_m) + j] = gicov[(i * grad_m) + j] * gicov[(i * grad_m) + j];
}



extern "C" float *ellipsematching_setup(int grad_m, int grad_n, float *host_grad_x, float *host_grad_y);

float *ellipsematching_setup(int grad_m, int grad_n, float *host_grad_x, float *host_grad_y) {
	int MaxR = MAX_RAD + 2;

	unsigned int grad_mem_size = sizeof(float) * grad_m * grad_n;

	// allocate device memory
	float *device_grad_x, *device_grad_y;
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_grad_x, grad_mem_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_grad_y, grad_mem_size));

	// copy host memory to device
	CUDA_SAFE_CALL( hipMemcpy( device_grad_x, host_grad_x, grad_mem_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( device_grad_y, host_grad_y, grad_mem_size, hipMemcpyHostToDevice) );
    
	// Bind input matrices to textures
    CUDA_SAFE_CALL( hipBindTexture(0, t_grad_x, device_grad_x, grad_mem_size));
    CUDA_SAFE_CALL( hipBindTexture(0, t_grad_y, device_grad_y, grad_mem_size));

	// allocate device memory for result
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_gicov, grad_mem_size));
	// initialize memory (some of the cells are never assigned a value in the kernel)
	CUDA_SAFE_CALL( hipMemset( device_gicov, 0, grad_mem_size) );

	// setup execution parameters
	dim3 grid( grad_n - (2 * MaxR), 1, 1);
	dim3 threads( grad_m - (2 * MaxR), 1, 1);
    
	// execute the kernel
	ellipsematching_kernel <<< grid, threads, 0 >>> (device_grad_x, device_grad_y, grad_m, grad_n, device_gicov);

	// check if kernel execution caused an error
	CUT_CHECK_ERROR("Kernel execution failed");

	// allocate mem for the result on host side
	float *host_gicov = (float *) malloc(grad_mem_size);
	// copy result from device to host
	CUDA_SAFE_CALL( hipMemcpy( host_gicov, device_gicov, grad_mem_size, hipMemcpyDeviceToHost) );

	// cleanup memory
	free(host_grad_x);
	free(host_grad_y);
	hipUnbindTexture(t_grad_x);
	hipUnbindTexture(t_grad_y);
	CUDA_SAFE_CALL(hipFree(device_grad_x));
	CUDA_SAFE_CALL(hipFree(device_grad_y));

	return host_gicov;
}




__global__ void dilate_f_kernel(float *img, int img_m, int img_n, int strel_m, int strel_n, float *dilated) {
	int i, j, el_i, el_j, x, y;
	int el_center_i = strel_m / 2, el_center_j = strel_n / 2;
	float max, temp;

	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	i = thread_id % img_m;
	j = thread_id / img_m;

	max = 0.0;

	for(el_i = 0; el_i < strel_m; el_i++) {
		y = i - el_center_i + el_i;
		if((y >= 0) && (y < img_m)) {
			for(el_j = 0; el_j < strel_n; el_j++) {
				x = j - el_center_j + el_j;
				if ((x >= 0) &&	(x < img_n) && (c_strel[(el_i * strel_n) + el_j] != 0)) {
					int addr = (x * img_m) + y;
					temp = tex1Dfetch(t_img, addr);
					if (temp > max) max = temp;
				}
			}
		}
	}

	dilated[(i * img_n) + j] = max;
}



extern "C" float *dilate_f_setup(int max_gicov_m, int max_gicov_n, int strel_m, int strel_n);

float *dilate_f_setup(int max_gicov_m, int max_gicov_n, int strel_m, int strel_n) {

	// Compute memory sizes
	unsigned int max_gicov_mem_size = sizeof(float) * max_gicov_m * max_gicov_n;

	// allocate device memory for result
	float* device_img_dilated;
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_img_dilated, max_gicov_mem_size) );
	
	CUDA_SAFE_CALL( hipBindTexture(0, t_img, device_gicov, max_gicov_mem_size));
    
	int num_threads = max_gicov_m * max_gicov_n;
	int threads_per_block = 176;
	int num_blocks = (int) (((float) num_threads / (float) threads_per_block) + 0.5);
    
	// setup execution parameters
	dim3 grid(num_blocks, 1, 1);
	dim3 threads(threads_per_block, 1, 1);

	// execute the kernel
	dilate_f_kernel <<< grid, threads, 0 >>> (device_gicov, max_gicov_m, max_gicov_n, strel_m, strel_n, device_img_dilated);

	// check if kernel execution generated an error
	CUT_CHECK_ERROR("Kernel execution failed");

	// allocate mem for the result on host side
	float *host_img_dilated = (float*) malloc(max_gicov_mem_size);
	// copy result from device to host
	CUDA_SAFE_CALL(hipMemcpy(host_img_dilated, device_img_dilated, max_gicov_mem_size, hipMemcpyDeviceToHost));

	// cleanup memory
	hipUnbindTexture(t_img);
	CUDA_SAFE_CALL(hipFree(device_gicov));
	CUDA_SAFE_CALL(hipFree(device_img_dilated));

	return host_img_dilated;
}


// Chooses the most appropriate GPU on which to execute
void select_device() {
	// Figure out how many devices exist
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	
	// Choose the device with the largest number of multiprocessors
	if (num_devices > 0) {
		int max_multiprocessors = 0, max_device = -1;
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
	}
	
	// The following is to remove the API initialization overhead from the runtime measurements
	hipFree(0);
}


void transfer_constants(float *host_sin_angle, float *host_cos_angle, int *host_tX, int *host_tY, int strel_m, int strel_n, float *host_strel) {
	// Compute the sizes of the matrices
	unsigned int angle_mem_size = sizeof(float) * NPOINTS;
	unsigned int t_mem_size = sizeof(int) * NCIRCLES * NPOINTS;
	unsigned int strel_mem_size = sizeof(float) * strel_m * strel_n;

	// Copy the matrices from host memory to device constant memory
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_sin_angle"), host_sin_angle, angle_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_cos_angle"), host_cos_angle, angle_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_tX"), host_tX, t_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_tY"), host_tY, t_mem_size, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_strel"), host_strel, strel_mem_size, 0, hipMemcpyHostToDevice));
}

